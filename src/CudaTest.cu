#include "hip/hip_runtime.h"
#include "KrimzLib.hpp"


// Function declarations
void cpuInterpolation(kl::color*, kl::size, kl::triangle);
__global__ void gpuInterpolation(kl::color*, kl::size, kl::triangle);// cpu => 491401

// Main func
int main() {
    // Rendering device type
    const bool useGpu = true;

    // Ram frame buffer
    kl::image cpuFrameBuffer(kl::size(1600, 900));

    // Triangle creation
    kl::triangle testTriangle;
    testTriangle.vertices[0].world = kl::vec3(100, cpuFrameBuffer.getHeight() - 100, 0);
    testTriangle.vertices[0].color = kl::constant::colors::red;
    testTriangle.vertices[1].world = kl::vec3(cpuFrameBuffer.getWidth() / 2, 100, 0);
    testTriangle.vertices[1].color = kl::constant::colors::green;
    testTriangle.vertices[2].world = kl::vec3(cpuFrameBuffer.getWidth() - 100, cpuFrameBuffer.getHeight() - 100, 0);
    testTriangle.vertices[2].color = kl::constant::colors::blue;

    // Triangle rendering
    kl::time::staticGetElapsed();
    if (useGpu) {
        kl::gpubuff<kl::color> gpuFrameBuffer(cpuFrameBuffer.getPixelCount());
        gpuInterpolation<<<cpuFrameBuffer.getPixelCount() / 256 + 1, 256>>>(gpuFrameBuffer.getPointer(), cpuFrameBuffer.getSize(), testTriangle);
        gpuFrameBuffer.toRAM(cpuFrameBuffer.getPointer());
    }
    else {
        cpuInterpolation(cpuFrameBuffer.getPointer(), cpuFrameBuffer.getSize(), testTriangle);
    }
    std::cout << (useGpu ? "Gpu " : "Cpu ") << " elapsed time: " << kl::time::staticGetElapsed() << std::endl;

    // Window start function definition
    kl::window testWindow;
    testWindow.windowStart = [&]() {
        testWindow.renderImage(cpuFrameBuffer);
    };

    // Window creation
    testWindow.startNew(cpuFrameBuffer.getSize(), "Test", false);
    return 0;
}


// Triangle interpolation with cpu
void cpuInterpolation(kl::color* frameBuffer, kl::size frameSize, kl::triangle tr) {
    tr.computeInterConsts();
    for(int i = 0; i < frameSize.width * frameSize.height; i++) {
        // Getting the x and y of the point
        const int x = i % frameSize.width;
        const int y = i / frameSize.width;

        // Calculating the interpolation weights
        kl::vec3 weights = tr.getInterWeights(kl::vec2(x, y));

        // Checking if the point is inside the triangle
        if (tr.inTriangle(weights)) {
            // Interpolating the color
            frameBuffer[i] = tr.interpolateColor(weights);
        }
        else {
            // Drawing the background
            frameBuffer[i] = kl::constant::colors::gray;
        }
    }
}

struct vec2 {
    double x;
    double y;
};

// Triangle interpolation kernel
__global__ void gpuInterpolation(kl::color* frameBuffer, kl::size frameSize, kl::triangle tr) {
    // Getting the thread index
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Checking if the i is inside the frame
    if (i < frameSize.width * frameSize.height) {
        // Getting the x and y of the point
        const int x = i % frameSize.width;
        const int y = i / frameSize.width;

        // Calculating the interpolation constants
        double tempConst = 1 / ((tr.vertices[1].world.y - tr.vertices[2].world.y) * (tr.vertices[0].world.x - tr.vertices[2].world.x) + (tr.vertices[2].world.x - tr.vertices[1].world.x) * (tr.vertices[0].world.y - tr.vertices[2].world.y));
        double w1xConst = (tr.vertices[1].world.y - tr.vertices[2].world.y) * tempConst;
        double w1yConst = (tr.vertices[2].world.x - tr.vertices[1].world.x) * tempConst;
        double w2xConst = (tr.vertices[2].world.y - tr.vertices[0].world.y) * tempConst;
        double w2yConst = (tr.vertices[0].world.x - tr.vertices[2].world.x) * tempConst;

        // Calculating the interpolation weights
        double interWeight1 = (x - tr.vertices[2].world.x) * w1xConst + (y - tr.vertices[2].world.y) * w1yConst;
        double interWeight2 = (x - tr.vertices[2].world.x) * w2xConst + (y - tr.vertices[2].world.y) * w2yConst;
        double interWeight3 = 1 - interWeight1 - interWeight2;

        // Checking if the point is inside the triangle
        if (!(interWeight1 < 0 || interWeight2 < 0 || interWeight3 < 0)) {
            // Interpolating the color
            frameBuffer[i].r = kl::byte(tr.vertices[0].color.r * interWeight1 + tr.vertices[1].color.r * interWeight2 + tr.vertices[2].color.r * interWeight3);
            frameBuffer[i].g = kl::byte(tr.vertices[0].color.g * interWeight1 + tr.vertices[1].color.g * interWeight2 + tr.vertices[2].color.g * interWeight3);
            frameBuffer[i].b = kl::byte(tr.vertices[0].color.b * interWeight1 + tr.vertices[1].color.b * interWeight2 + tr.vertices[2].color.b * interWeight3);
        }
        else {
            // Drawing the background
            frameBuffer[i].r = 50;
            frameBuffer[i].g = 50;
            frameBuffer[i].b = 50;
        }
    }
}
