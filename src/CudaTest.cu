#include "hip/hip_runtime.h"
#include "KrimzLib.hpp"


// Function declarations
__global__ void gpuInterpolation(kl::color*, kl::size, kl::triangle);

// Main func
int main() {
    // Frame buffer generation
    kl::image cpuFrameBuffer(kl::size(1600, 900));
    kl::gpubuff<kl::color> gpuFrameBuffer(cpuFrameBuffer.getPixelCount());

    // Triangle creation
    kl::triangle testTriangle;
    testTriangle.vertices[0].world = kl::vec3(100, cpuFrameBuffer.getHeight() - 100, 0);
    testTriangle.vertices[0].color = kl::color(255, 0, 0);
    testTriangle.vertices[1].world = kl::vec3(cpuFrameBuffer.getWidth() / 2, 100, 0);
    testTriangle.vertices[1].color = kl::color(0, 255, 0);
    testTriangle.vertices[2].world = kl::vec3(cpuFrameBuffer.getWidth() - 100, cpuFrameBuffer.getHeight() - 100, 0);
    testTriangle.vertices[2].color = kl::color(0, 0, 255);

    // Triangle rendering
    kl::time::staticGetElapsed();
    gpuInterpolation<<<cpuFrameBuffer.getPixelCount() / 256 + 1, 256>>>(gpuFrameBuffer.getPointer(), cpuFrameBuffer.getSize(), testTriangle);
    hipDeviceSynchronize();
    gpuFrameBuffer.toRAM(cpuFrameBuffer.getPointer());
    std::cout << "Gpu elapsed time: " << kl::time::staticGetElapsed() << std::endl;

    // Window start function definition
    kl::window testWindow;
    testWindow.windowStart = [&]() {
        testWindow.renderImage(cpuFrameBuffer);
    };

    // Window creation
    testWindow.startNew(cpuFrameBuffer.getSize(), "Test", false);
    return 0;
}

// Triangle interpolation kernel
__global__ void gpuInterpolation(kl::color* frameBuffer, kl::size frameSize, kl::triangle tr) {
    // Getting the thread index
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Checking if the i is inside the frame
    if (i < frameSize.width * frameSize.height) {
        // Getting the x and y of the point
        const int x = i % frameSize.width;
        const int y = i / frameSize.width;

        // Calculating the interpolation constants
        float tempConst = 1 / ((tr.vertices[1].world.y - tr.vertices[2].world.y) * (tr.vertices[0].world.x - tr.vertices[2].world.x) + (tr.vertices[2].world.x - tr.vertices[1].world.x) * (tr.vertices[0].world.y - tr.vertices[2].world.y));
        float w1xConst = (tr.vertices[1].world.y - tr.vertices[2].world.y) * tempConst;
        float w1yConst = (tr.vertices[2].world.x - tr.vertices[1].world.x) * tempConst;
        float w2xConst = (tr.vertices[2].world.y - tr.vertices[0].world.y) * tempConst;
        float w2yConst = (tr.vertices[0].world.x - tr.vertices[2].world.x) * tempConst;

        // Calculating the interpolation weights
        float interWeight1 = (x - tr.vertices[2].world.x) * w1xConst + (y - tr.vertices[2].world.y) * w1yConst;
        float interWeight2 = (x - tr.vertices[2].world.x) * w2xConst + (y - tr.vertices[2].world.y) * w2yConst;
        float interWeight3 = 1 - interWeight1 - interWeight2;

        // Checking if the point is inside the triangle
        if (!(interWeight1 < 0 || interWeight2 < 0 || interWeight3 < 0)) {
            // Interpolating the color
            frameBuffer[i].r = kl::byte(tr.vertices[0].color.r * interWeight1 + tr.vertices[1].color.r * interWeight2 + tr.vertices[2].color.r * interWeight3);
            frameBuffer[i].g = kl::byte(tr.vertices[0].color.g * interWeight1 + tr.vertices[1].color.g * interWeight2 + tr.vertices[2].color.g * interWeight3);
            frameBuffer[i].b = kl::byte(tr.vertices[0].color.b * interWeight1 + tr.vertices[1].color.b * interWeight2 + tr.vertices[2].color.b * interWeight3);
        }
        else {
            // Drawing the background
            frameBuffer[i].r = 50;
            frameBuffer[i].g = 50;
            frameBuffer[i].b = 50;
        }
    }
}
