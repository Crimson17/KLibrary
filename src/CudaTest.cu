#include "hip/hip_runtime.h"
#include "KrimzLib.hpp"


__global__ void testKernel(kl::vec2* dataArray) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    dataArray[i].x = i;
    dataArray[i].y = i * 3.14;
}

int main() {
    const int n = 6;

    kl::array<kl::vec2> a(n);
    kl::gpubuff<kl::vec2> b(n);

    testKernel<<<1, n>>>(b.getPointer());
    hipDeviceSynchronize();

    b.toRAM(a.getPointer());

    for(int i = 0; i < n; i++) {
        a[i].print();
    }
    
    return 0;
}
