#include "hip/hip_runtime.h"
#include "KrimzLib.h"


__global__ void testKernel(kl::vec2* dataArray) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    dataArray[i].x = i;
    dataArray[i].y = i * 3.14;
}

int main() {
    const int n = 6;

    kl::vec2* a = new kl::vec2[n];
    kl::gpubuff<kl::vec2> b(n);

    testKernel<<<1, n>>>(b.getPointer());
    hipDeviceSynchronize();

    b.toRAM(a);

    for(int i = 0; i < n; i++) {
        a[i].print();
    }

    delete[] a;

    return 0;
}
